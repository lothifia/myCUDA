#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <hipblas.h>
#include "myhead.h"
// #define BLOCKSIZE 32
template <const int BM, const int BN, const int BK, const int TM, const int TN>
__global__ void __launch_bounds__((BM * BN) / (TM * TN), 1)
mul_shared_mem_2d( float* A,  float* B, float* C, int M, int K, int N, float alpha, float beta) {


    const uint block_row = blockIdx.y; // block row
    const uint block_col = blockIdx.x; // block col    WTFFFFFFFFF?
    const uint thread_Row = threadIdx.x / (BN / TN); // thread
    const uint thread_Col = threadIdx.x % (BN / TN /* BN / TN -> thread"block"*/);
    __shared__ float s_A[BM * BK];
    __shared__ float s_B[BK * BN];
    float thread_TM[TM * TN] = {0.0};

    const uint totalResPerBlock = BM * BN;
    const uint numThreadBlocktile = totalResPerBlock / (TM * TN);
    // printf("totalResPerBlock %d numThreadBlocktile %d \n", totalResPerBlock, numThreadBlocktile);
    /*per block is a (16 * 16) size block with threads, in fact is a 256 vector, we need to use this 256 thread deal 128 * 8 size data,
    means per thread execute 128 * 8 / 256 = 4 data. the numThreadBlocktile means 256 thread .
    nTBt/ BK = 256 / 8 = 32 means each threads the gap between data per thread exec in A.
    same on B. 
    example in A: 
    |1 the data map to block  ********
    |2 the data map to block  ********
    | .......
    |32 the data map to block ********
    |the data cope in loop    --------
    |the data cope in loop    --------
    |the data cope in loop    --------
    |the data cope in loop    --------
    */
    const uint innerAx = threadIdx.x / (BK / 4); // BK / 4 -> 8 / 4 means 4 data per thread - > 2 set per row(8 means 8 data per row)
    const uint innerAy = threadIdx.x % (BK / 4);
    const uint innerBx = threadIdx.x / (BN / 4);
    const uint innerBy = threadIdx.x % (BN / 4);

    // printf("strideA %d innerAx %d innerAy %d strideB %d innerBx %d innerBy %d \n", strideA, innerAx, innerAy, strideB, innerBx, innerBy);
    float regM[TM] = {0.0};
    float regN[TN] = {0.0};
    A += block_row * BM * K;
    B += block_col * BN;
    C += block_row * BM * N + block_col * BN;

    for(uint block_idx = 0; block_idx < K; block_idx += BK) {
        // for(int row_offest = 0; row_offest < BM; row_offest += strideA) {
        //     s_A[innerAy + (innerAx + row_offest) * BK] = 
        //         A[innerAy + (innerAx + row_offest) * K];
        // }
        // for(int row_offest = 0; row_offest < BK; row_offest += strideB) {
        //     s_B[innerBy + (innerBx + row_offest) * BN] =
        //         B[innerBy + (innerBx + row_offest) * N];
        // }
        // populate the SMEM caches
        float4 tmp = reinterpret_cast<const float4*>(&A[innerAx * K + innerAy * 4])[0];
        s_A[innerAx + (innerAy * 4 + 0) * BM] = tmp.x;
        s_A[innerAx + (innerAy * 4 + 1) * BM] = tmp.y;
        s_A[innerAx + (innerAy * 4 + 2) * BM] = tmp.z;
        s_A[innerAx + (innerAy * 4 + 3) * BM] = tmp.w;
        reinterpret_cast<float4 *>(&s_B[innerBx * BN + innerBy * 4])[0] =
          reinterpret_cast<float4 *>(&B[innerBx * N + innerBy * 4])[0];

        __syncthreads();  

        A += BK;
        B += BK * N;
        /*each thread should exec 8 * 8 = 64 data. so there is a regM and regN , per out loop deal 64 times data single fma*/
        for(uint idx_BK = 0; idx_BK < BK; ++idx_BK) {
            for(uint j = 0; j < TM; ++j) {
                regM[j] = s_A[idx_BK * BM + TM * thread_Row + j];
            }
            for(uint j = 0; j < TN; ++j) {
                regN[j] = s_B[idx_BK * BN + TN * thread_Col + j];       
            }
            for(uint i = 0; i < TM; ++i) {
                for(uint j = 0; j < TN; ++j) {
                    thread_TM[i * TN + j] += regM[i] * regN[j];
                }
            }
        }
        __syncthreads();
    }

    // for(uint i = 0; i < TM; ++i) {
    //     for(uint j = 0; j < TN; ++j) {
    //         // C[(thread_Row * TM + i) * N + thread_Col * TN + j] = thread_TM[i * TN + j];
    //         C[(thread_Row * TM + i) * N + thread_Col * TN + j] = alpha * thread_TM[i * TN + j] + beta * C[(thread_Row * TM + i) * N + thread_Col * TN + j];
    //     }
    // }
    for(int resIdxM = 0; resIdxM < TM; ++resIdxM) {
        for(int resIdxN = 0; resIdxN < TN; resIdxN += 4) {
          float4 tmp = reinterpret_cast<float4 *>(&C[(thread_Row * TM + resIdxM) * N + thread_Col * TN + resIdxN])[0];
          tmp.x = alpha * thread_TM[resIdxM * TN + resIdxN] + beta * tmp.x;
          tmp.y = alpha * thread_TM[resIdxM * TN + resIdxN + 1] + beta * tmp.y; 
          tmp.z = alpha * thread_TM[resIdxM * TN + resIdxN + 2] + beta * tmp.z;
          tmp.w = alpha * thread_TM[resIdxM * TN + resIdxN + 3] + beta * tmp.w;
          reinterpret_cast<float4 *>(&C[(thread_Row * TM + resIdxM) * N + thread_Col * TN + resIdxN])[0] = tmp;
        }
    }
}

int main() {
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp,dev));
    printf("Using device %d: %s %d\n",dev,deviceProp.name, deviceProp.warpSize);
    printf("Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
    CHECK(hipSetDevice(dev));

    size_t nx_A = 1 << 13;
    // size_t nx_B = 1 << 13;
    size_t ny_A = 1 << 13;
    // size_t ny_B = 1 << 13;
    size_t nxy_A = nx_A * ny_A; 
    size_t nByte_A = nx_A * ny_A * sizeof(float);
    // size_t totByte_B = nx_B * ny_B * sizeof(float);
    const int TM = 8;
    const int TN = 8;
    const int BK = 8;
    const int BM = 128;
    const int BN = 128;
    dim3 gridDim(CEIL_DIV(nx_A, BM), CEIL_DIV(ny_A, BN));
    dim3 blockDim((BM * BN) / (TM * TN));
    float elapsed_time;
    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);

    float* h_A = (float* )malloc(nByte_A);
    float* h_B = (float* )malloc(nByte_A);
    float* h_C = (float* )malloc(nByte_A);
    float* d_A = NULL;
    float* d_B = NULL;
    float* d_C = NULL;
    CHECK(hipMalloc((void**)&d_A, nByte_A));
    CHECK(hipMalloc((void**)&d_B, nByte_A));
    CHECK(hipMalloc((void**)&d_C, nByte_A));
    for(int i = 0; i < nxy_A; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }
    double sTime = cpuSecond();
    CHECK(hipMemcpy(d_A, h_A, nByte_A, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nByte_A, hipMemcpyHostToDevice));

    hipblasHandle_t handle;
    hipblasCreate(&handle); 
    float alpha = 1.0f;
    float beta = 0.0f; 
    // hipFuncAttribute(mul_shared_mem<32>, hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxShared);
    hipEventRecord(beg);
    mul_shared_mem_2d<BM, BN, BK, TM, TN><<<gridDim, blockDim>>>(d_A, d_B, d_C, nx_A, ny_A, nx_A, alpha, beta);
    // hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nx_A, ny_A, nx_A, &alpha, d_A, nx_A, d_B, nx_A, &beta, d_C, nx_A);
    // sgemm2DBlocktiling<BM, BN, BK, TM, TN><<<gridDim, blockDim>>> (nx_A, ny_A, nx_A, 1.0f, d_A, d_B, 0.0f, d_C);
    hipEventRecord(end);
    hipEventSynchronize(beg);
    hipEventSynchronize(end); 
    hipEventElapsedTime(&elapsed_time, beg, end);
    elapsed_time /= 1.0e3f;
    long flops = 2 * nx_A * ny_A * nx_A;
    printf("elapsed time %f s, TFLOPS %f\n", elapsed_time, flops / elapsed_time / 1.0e12);

    CHECK(hipMemcpy(h_C, d_C, nByte_A, hipMemcpyDeviceToHost));
    CHECK(hipDeviceSynchronize());
    printf("over \n");
    // for(int i = 0; i < nxy_A / nx_A; i++) {
    //     printf("%f ", h_C[i]);
    //     if(i % nx_A == nx_A - 1) {
    //         printf("\n");
    //     }
    // }
    // for(int i = 0; i < nxy_A; i++) {
    //     printf("%f ", h_A[i]);
    //     if(i % nx_A == nx_A - 1) {
    //         printf("\n");
    //     }
    // }
    // for(int i = 0; i < nxy_A; i++) {
    //     printf("%f ", h_B[i]);
    //     if(i % nx_A == nx_A - 1) {
    //         printf("\n");
    //     }
    // }
    return 0;
}