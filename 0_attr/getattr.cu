#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
int main() {
    hipDeviceProp_t prop;
    int device;
    
    // 获取当前设备号
    hipGetDevice(&device);
    
    // 获取当前设备的属性
    hipGetDeviceProperties(&prop, device);

    printf("设备名称: %s\n", prop.name);
    printf("计算能力: %d.%d\n", prop.major, prop.minor);
    printf("每个线程块最大线程数: %d\n", prop.maxThreadsPerBlock);
    printf("线程块x方向最大线程数: %d\n", prop.maxThreadsDim[0]);
    printf("线程块y方向最大线程数: %d\n", prop.maxThreadsDim[1]);
    printf("线程块z方向最大线程数: %d\n", prop.maxThreadsDim[2]);
    printf("reg per thread: %d\n", prop.regsPerBlock / prop.maxThreadsPerBlock);
    printf("reg per Block: %d\n", prop.regsPerBlock);
    printf("Smem per Block: %zu\n", prop.sharedMemPerBlock);
    printf("threads per Block: %d\n", prop.maxThreadsPerBlock);
    printf("shared mem per sm: %zu B\n", prop.sharedMemPerMultiprocessor);
 
    return 0;

}
