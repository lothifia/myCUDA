#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
void CalTheoreticalBandWidth()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);               // 获取设备上的GPU个数

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);    // 获取当前GPU的相关属性


        std::cout << "GPU: " << i << std::endl;
        std::cout << "Name: " << deviceProp.name << std::endl;
        std::cout << "Bit width: " << deviceProp.memoryBusWidth << " bit" << std::endl;
        std::cout << "Memory clock rate: " << deviceProp.memoryClockRate << " kHz" << std::endl;

        int bw = static_cast<size_t>(deviceProp.memoryClockRate) * 1000 * deviceProp.memoryBusWidth / 8 * 2 / 1000000000;
        
        std::cout << "Theoretical band width = " << bw << " GB/s" << std::endl;
    }
}

int main() {
    CalTheoreticalBandWidth();
    hipDeviceProp_t prop;
    int device;
    
    // 获取当前设备号
    hipGetDevice(&device);
    
    // 获取当前设备的属性
    hipGetDeviceProperties(&prop, device);

    printf("设备名称: %s\n", prop.name);
    printf("计算能力: %d.%d\n", prop.major, prop.minor);
    printf("每个线程块最大线程数: %d\n", prop.maxThreadsPerBlock);
    printf("线程块x方向最大线程数: %d\n", prop.maxThreadsDim[0]);
    printf("线程块y方向最大线程数: %d\n", prop.maxThreadsDim[1]);
    printf("线程块z方向最大线程数: %d\n", prop.maxThreadsDim[2]);
    printf("reg per thread: %d\n", prop.regsPerBlock / prop.maxThreadsPerBlock);
    printf("reg per Block: %d\n", prop.regsPerBlock);
    printf("Smem per Block: %zu\n", prop.sharedMemPerBlock);
    printf("threads per Block: %d\n", prop.maxThreadsPerBlock);
    printf("shared mem per sm: %zu B\n", prop.sharedMemPerMultiprocessor);
    // printf("", prop.reg)



    hipDeviceProp_t props = prop;
    printf("Device ID: %d\n\
        Name: %s\n\
        Compute Capability: %d.%d\n\
        memoryBusWidth: %d\n\
        maxThreadsPerBlock: %d\n\
        maxThreadsPerMultiProcessor: %d\n\
        maxRegsPerBlock: %d\n\
        maxRegsPerMultiProcessor: %d\n\
        totalGlobalMem: %zuMB\n\
        sharedMemPerBlock: %zuKB\n\
        sharedMemPerMultiprocessor: %zuKB\n\
        totalConstMem: %zuKB\n\
        multiProcessorCount: %d\n\
        Warp Size: %d\n",
             0, props.name, props.major, props.minor, props.memoryBusWidth,
             props.maxThreadsPerBlock, props.maxThreadsPerMultiProcessor,
             props.regsPerBlock, props.regsPerMultiprocessor,
             props.totalGlobalMem / 1024 / 1024, props.sharedMemPerBlock / 1024,
             props.sharedMemPerMultiprocessor / 1024, props.totalConstMem / 1024,
             props.multiProcessorCount, props.warpSize);
    return 0;

}
