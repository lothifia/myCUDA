#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include "myhead.h"
// #define BLOCKSIZE 32
template <const int AM, const int AK, const int BN, const int tmp>
__global__ void mul_shared_mem_1d(float* d_A, float* d_B, float* d_C, size_t nx, size_t nk, size_t ny) {
    size_t cCol = blockIdx.y; // block.y
    size_t cRow = blockIdx.x; // block.x
    size_t tCol = threadIdx.x % BN; // 1. row first ! 2. "mapping a one-dimensional index to a two-dimensional coordinate"
    size_t tRow = threadIdx.x / BN; // same on

    d_A += cRow * AK * nk; // the block first row
    d_B += cCol * BN; // the block first col
    d_C += cRow * BN * ny + cCol * BN; // the target block fisrt thread

    __shared__ float s_A[AM * AK];
    __shared__ float s_B[AK * BN];

    assert(AM * AK == blockDim.x);
    assert(AK * BN == blockDim.x);
    const int innerAx = threadIdx.x / AK;
    const int innerAy = threadIdx.x % AK;
    const int innerBx = threadIdx.x / BN;
    const int innerBy = threadIdx.x % BN;
    float threadTmp[tmp]{0.0};
    for(int block_i = 0; block_i < nk; block_i += AK) {       
        s_A[innerAx * AK + innerAy] = d_A[innerAx * nk + innerAy];
        s_B[innerBx * BN + innerBy] = d_B[innerBx * ny + innerBy];
        __syncthreads();
        d_A += AK; // next block in A
        d_B += AK * ny; // next block in B
        for(int i = 0; i < AK; ++ i) {
            float tmpB = s_B[i * BN + innerBy];
            for (int j = 0; j < tmp; ++ j) {
                threadTmp[j] += s_A[(tRow * tmp + j ) * AK + i] * tmpB;
            }
        }
        __syncthreads();
    }
    for (int i = 0; i < tmp; ++ i) {
        d_C[(tRow * tmp + i) * ny + tCol] = threadTmp[i];
    }
}
template <const int BLOCKSIZE>
__global__ void mul_shared_mem(float* d_A, float* d_B, float* d_C, size_t nx, size_t nk, size_t ny) {
    size_t cCol = blockIdx.y; // block.y
    size_t cRow = blockIdx.x; // block.x
    size_t tCol = threadIdx.x % BLOCKSIZE; // 1. row first ! 2. "mapping a one-dimensional index to a two-dimensional coordinate"
    size_t tRow = threadIdx.x / BLOCKSIZE; // same on

    d_A += cRow * BLOCKSIZE * nk; // the block first row
    d_B += cCol * BLOCKSIZE; // the block first col
    d_C += cRow * BLOCKSIZE * ny + cCol * BLOCKSIZE; // the target block fisrt thread

    __shared__ float s_A[BLOCKSIZE * BLOCKSIZE];
    __shared__ float s_B[BLOCKSIZE * BLOCKSIZE];


    float tmp = 0.0;
    for(int block_i = 0; block_i < nk; block_i += BLOCKSIZE) {       

        s_A[tRow * BLOCKSIZE + tCol] = d_A[tRow * nk + tCol];
        s_B[tRow * BLOCKSIZE + tCol] = d_B[tRow * ny + tCol];
        __syncthreads();
        d_A += BLOCKSIZE; // next block in A
        d_B += BLOCKSIZE * ny; // next block in B
        
        
        for(int i = 0; i < BLOCKSIZE; ++ i) {
            tmp += s_A[tRow * BLOCKSIZE + i] * s_B[i * BLOCKSIZE + tCol];
        }
        __syncthreads();
    }
    d_C[tRow * ny + tCol] = tmp; 
    /*
         d_C pointer at the start of block.
         * If in (tRow = 0, tCol = 0) d_C = tmp;
         * if (1, 1) then d_C have cross the WHOLE line to get the next row and plus col to get the thread.
    */
}


int main() {
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp,dev));
    printf("Using device %d: %s %d\n",dev,deviceProp.name, deviceProp.warpSize);
    printf("Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
    CHECK(hipSetDevice(dev));

    size_t nx_A = 1 << 13;
    // size_t nx_B = 1 << 13;
    size_t ny_A = 1 << 13;
    // size_t ny_B = 1 << 13;
    size_t nxy_A = nx_A * ny_A; 
    size_t nByte_A = nx_A * ny_A * sizeof(float);
    // size_t totByte_B = nx_B * ny_B * sizeof(float);
    const int AM = 64;
    const int AK = 8;
    const int BN = 64;
    const int tmp = 8;
    
    dim3 gridDim(CEIL_DIV(nx_A, AM), CEIL_DIV(ny_A, BN));
    dim3 blockDim((AM * BN) / tmp);

    float* h_A = (float* )malloc(nByte_A);
    float* h_B = (float* )malloc(nByte_A);
    float* h_C = (float* )malloc(nByte_A);
    float* d_A = NULL;
    float* d_B = NULL;
    float* d_C = NULL;
    CHECK(hipMalloc((void**)&d_A, nByte_A));
    CHECK(hipMalloc((void**)&d_B, nByte_A));
    CHECK(hipMalloc((void**)&d_C, nByte_A));
    

    for(int i = 0; i < nxy_A; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }
    double sTime = cpuSecond();
    CHECK(hipMemcpy(d_A, h_A, nByte_A, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nByte_A, hipMemcpyHostToDevice));
    printf("mem copy time %f \n", cpuSecond() - sTime);

    // hipFuncAttribute(mul_shared_mem<32>, hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxShared);
    // mul_shared_mem<32><<<gridDim, blockDim>>> (d_A, d_B, d_C, nx_A, nx_A, ny_A);
    mul_shared_mem_1d<AM, AK, BN, tmp><<<gridDim, blockDim>>> (d_A, d_B, d_C, nx_A, nx_A, ny_A);
    CHECK(hipMemcpy(h_C, d_C, nByte_A, hipMemcpyDeviceToHost));
    CHECK(hipDeviceSynchronize());
    printf("total dev time %f \n", cpuSecond() - sTime);
    printf("over \n");
    for(int i = 0; i < nxy_A / nx_A; i++) {
        printf("%f ", h_C[i]);
        if(i % nx_A == nx_A - 1) {
            printf("\n");
        }
    }
    // for(int i = 0; i < nxy_A; i++) {
    //     printf("%f ", h_A[i]);
    //     if(i % nx_A == nx_A - 1) {
    //         printf("\n");
    //     }
    // }
    // for(int i = 0; i < nxy_A; i++) {
    //     printf("%f ", h_B[i]);
    //     if(i % nx_A == nx_A - 1) {
    //         printf("\n");
    //     }
    // }
    return 0;
}